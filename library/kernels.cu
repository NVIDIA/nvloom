#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2024-2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "kernels.cuh"
#include "error.h"
#include "nvloom.h"
#include <hiprand/hiprand_kernel.h>

const unsigned int numThreadPerBlock = 512;

template<typename T>
using write_to_memory = void (*)(T*, T);

template<typename T>
using reduce_from_memory = void (*)(T*, T*);

template<typename T>
__device__ void write_to_regular_memory(T *dst, T val) {
    *dst = val;
}

template<typename T>
__device__ void write_to_multicast_memory(T *dst, T val) {
#if __CUDA_ARCH__ >= 900
    static_assert(sizeof(T) == 4, "");
    asm ("multimem.st.weak.global.b32 [%0], %1;" : : "l"(dst), "r"(val) : "memory" );
#endif
}

template<typename T>
__device__ void reduce_from_multicast_ld_reduce(T *dst, T *val) {
#if __CUDA_ARCH__ >= 900
    static_assert(sizeof(T) == 4, "");
    uint result;
    asm ("multimem.ld_reduce.weak.global.add.u32 %0, [%1];" : "=r"(result) : "l"(val) : "memory");
    *dst = result;
#endif
}

template<typename T>
__device__ void reduce_from_multicast_red(T *dst, T *val) {
#if __CUDA_ARCH__ >= 900
    static_assert(sizeof(T) == 4, "");
    asm ("multimem.red.global.add.u32 [%0], %1;" : "+l"(dst) : "r"(*val) : "memory");
#endif
}

template<typename T, write_to_memory<T> write>
__global__ void stridingMemcpyKernel(unsigned int totalThreadCount, unsigned long long loopCount, T* dst, T* src, size_t sizeInElement) {
    T *dstEnd = dst + sizeInElement;
    size_t chunkSizeInElement = sizeInElement / totalThreadCount;

    size_t globalThreadId = blockDim.x * blockIdx.x + threadIdx.x;
    dst += globalThreadId;
    src += globalThreadId;

    // Calculate where to end the big pipelined copy
    size_t bigChunkSizeInElement = chunkSizeInElement / 12;
    T *dstBigEnd = dst + (bigChunkSizeInElement * 12) * totalThreadCount;

    for (unsigned int i = 0; i < loopCount; i++) {
        T* cdst = dst;
        T* csrc = src;

        while (cdst < dstBigEnd) {
            T pipe_0 = *csrc; csrc += totalThreadCount;
            T pipe_1 = *csrc; csrc += totalThreadCount;
            T pipe_2 = *csrc; csrc += totalThreadCount;
            T pipe_3 = *csrc; csrc += totalThreadCount;
            T pipe_4 = *csrc; csrc += totalThreadCount;
            T pipe_5 = *csrc; csrc += totalThreadCount;
            T pipe_6 = *csrc; csrc += totalThreadCount;
            T pipe_7 = *csrc; csrc += totalThreadCount;
            T pipe_8 = *csrc; csrc += totalThreadCount;
            T pipe_9 = *csrc; csrc += totalThreadCount;
            T pipe_10 = *csrc; csrc += totalThreadCount;
            T pipe_11 = *csrc; csrc += totalThreadCount;

            write(cdst, pipe_0); cdst += totalThreadCount;
            write(cdst, pipe_1); cdst += totalThreadCount;
            write(cdst, pipe_2); cdst += totalThreadCount;
            write(cdst, pipe_3); cdst += totalThreadCount;
            write(cdst, pipe_4); cdst += totalThreadCount;
            write(cdst, pipe_5); cdst += totalThreadCount;
            write(cdst, pipe_6); cdst += totalThreadCount;
            write(cdst, pipe_7); cdst += totalThreadCount;
            write(cdst, pipe_8); cdst += totalThreadCount;
            write(cdst, pipe_9); cdst += totalThreadCount;
            write(cdst, pipe_10); cdst += totalThreadCount;
            write(cdst, pipe_11); cdst += totalThreadCount;
        }

        // Take care of copies that didn't get aligned properly
        while (cdst < dstEnd) {
            write(cdst, *csrc); cdst += totalThreadCount; csrc += totalThreadCount;
        }
    }
}

template<typename T, reduce_from_memory<T> write>
__global__ void simpleMemcpyKernel(unsigned int totalThreadCount, unsigned long long loopCount, T* dst, T* src, size_t sizeInElement) {
    T *dstEnd = dst + sizeInElement;

    size_t globalThreadId = blockDim.x * blockIdx.x + threadIdx.x;
    dst += globalThreadId;
    src += globalThreadId;

    for (unsigned int i = 0; i < loopCount; i++) {
        T* cdst = dst;
        T* csrc = src;

        while (cdst < dstEnd) {
            write(cdst, csrc); cdst += totalThreadCount; csrc += totalThreadCount;
        }
    }
}

template<typename T, auto kernel>
void launchCopyKernel(hipDeviceptr_t dstBuffer, hipDeviceptr_t srcBuffer, size_t size, hipStream_t stream, unsigned long long loopCount) {
    unsigned int totalThreadCount = NvLoom::getLocalMultiprocessorCount() * numThreadPerBlock;

    // adjust size to elements (size is multiple of MB, so no truncation here)
    size_t sizeInElement = size / sizeof(T);

    dim3 gridDim(NvLoom::getLocalMultiprocessorCount(), 1, 1);
    dim3 blockDim(numThreadPerBlock, 1, 1);
    kernel <<<gridDim, blockDim, 0, stream>>> (totalThreadCount, loopCount, (T *) dstBuffer, (T *) srcBuffer, sizeInElement);
    CUDA_ASSERT(hipPeekAtLastError());
}

void copyKernel(hipDeviceptr_t dstBuffer, hipDeviceptr_t srcBuffer, size_t size, hipStream_t stream, unsigned long long loopCount) {
    launchCopyKernel<uint4, stridingMemcpyKernel<uint4, write_to_regular_memory> >(dstBuffer, srcBuffer, size, stream, loopCount);
}

void copyKernelMulticast(hipDeviceptr_t dstBuffer, hipDeviceptr_t srcBuffer, size_t size, hipStream_t stream, unsigned long long loopCount) {
    launchCopyKernel<uint, stridingMemcpyKernel<uint, write_to_multicast_memory> >(dstBuffer, srcBuffer, size, stream, loopCount);
}

void copyKernelMulticastLdReduce(hipDeviceptr_t dstBuffer, hipDeviceptr_t srcBuffer, size_t size, hipStream_t stream, unsigned long long loopCount) {
    launchCopyKernel<uint, simpleMemcpyKernel<uint, reduce_from_multicast_ld_reduce<uint> > >(dstBuffer, srcBuffer, size, stream, loopCount);
}

void copyKernelMulticastRed(hipDeviceptr_t dstBuffer, hipDeviceptr_t srcBuffer, size_t size, hipStream_t stream, unsigned long long loopCount) {
    launchCopyKernel<uint, simpleMemcpyKernel<uint, reduce_from_multicast_red<uint> > >(dstBuffer, srcBuffer, size, stream, loopCount);
}

__global__ void spinKernelDeviceMultistage(volatile int *latch1, volatile int *latch2, const unsigned long long timeoutClocks) {
    if (latch1) {
        register unsigned long long endTime = clock64() + timeoutClocks;
        while (!*latch1) {
            if (timeoutClocks != ~0ULL && clock64() > endTime) {
                return;
            }
        }

        *latch2 = 1;
    }

    register unsigned long long endTime = clock64() + timeoutClocks;
    while (!*latch2) {
        if (timeoutClocks != ~0ULL && clock64() > endTime) {
            break;
        }
    }
}

// Implement a 2-stage spin kernel for multi-node synchronization.
// One of the host nodes releases the first latch. Subsequently,
// the second latch is released, that is polled by all other devices
// latch1 argument is optional. If defined, kernel will spin on it until released, and then will release latch2.
// latch2 argument is mandatory. Kernel will spin on it until released.
// timeoutMs argument applies to each stage separately.
// However, since each kernel will spin on only one stage, total runtime is still limited by timeoutMs
hipError_t spinKernelMultistage(volatile int *latch1, volatile int *latch2, hipStream_t stream, unsigned long long timeoutMs) {
    ASSERT(latch2 != nullptr);

    unsigned long long timeoutClocks = NvLoom::getLocalClockRate() * timeoutMs;
    spinKernelDeviceMultistage<<<1, 1, 0, stream>>>(latch1, latch2, timeoutClocks);
    CUDA_ASSERT(hipPeekAtLastError());

    return hipSuccess;
}

__global__ void patternFillKernel(uint* dst, int seed, size_t bufferSize, int groupId, int groupSize) {
    unsigned long long threadId = blockDim.x * blockIdx.x + threadIdx.x;
    size_t totalThreadCount = gridDim.x * blockDim.x;
    char* dstEnd = ((char *) dst) + bufferSize;
    dst += threadId;

    hiprandStateXORWOW_t state;
    hiprand_init(seed, 0, threadId, &state);

    for (int i = 0; i < groupId; i++) {
        hiprand(&state);
    }

    while ((char *) dst < dstEnd) {
        *dst = hiprand(&state);
        dst += totalThreadCount;

        for (int i = 0; i < groupSize - 1; i++) {
            hiprand(&state);
        }
    }
}

void memsetBuffer(void *ptr, int seed, size_t size, hipStream_t stream, int groupId, int groupSize) {
    dim3 gridDim(NvLoom::getLocalMultiprocessorCount(), 1, 1);
    dim3 blockDim(numThreadPerBlock, 1, 1);
    patternFillKernel<<<gridDim, blockDim, 0, stream>>>((uint *)ptr, seed, size, groupId, groupSize);
    CUDA_ASSERT(hipPeekAtLastError());
}

void zeroOutBuffer(void *ptr, size_t size, hipStream_t stream) {
    CU_ASSERT(hipMemsetD8Async((hipDeviceptr_t) ptr, 0, size, stream));
}

void memsetBuffer(void *ptr, int seed, size_t size, hipStream_t stream, CopyType copyType, MemoryPurpose memoryPurpose) {
    if (copyType == COPY_TYPE_MULTICAST_LD_REDUCE) {
        memsetBuffer(ptr, seed, size, stream, MPIWrapper::getWorldRank(), MPIWrapper::getWorldSize());
    } else if (copyType == COPY_TYPE_MULTICAST_RED_ALL) {
        if (memoryPurpose == MemoryPurpose::MEMORY_SOURCE) {
            memsetBuffer(ptr, seed, size, stream, MPIWrapper::getWorldRank(), MPIWrapper::getWorldSize());
        } else {
            zeroOutBuffer(ptr, size, stream);
        }
    } else if (copyType == COPY_TYPE_MULTICAST_RED_SINGLE) {
        if (memoryPurpose == MemoryPurpose::MEMORY_SOURCE) {
            memsetBuffer(ptr, seed, size, stream, 0, 1);
        } else {
            zeroOutBuffer(ptr, size, stream);
        }
    } else {
        memsetBuffer(ptr, seed, size, stream, 0, 1);
    }
}

__global__ void patternCheckKernel(uint* buffer, int seed, size_t bufferSize, unsigned long long *errorCount, int groupSize, int multiplier) {
    uint* originalBuffer = buffer;
    unsigned long long threadId = blockDim.x * blockIdx.x + threadIdx.x;
    size_t totalThreadCount = gridDim.x * blockDim.x;
    char* bufferEnd = ((char *) buffer) + bufferSize;
    buffer += threadId;

    hiprandStateXORWOW_t state;
    hiprand_init(seed, 0, threadId, &state);

    while ((char *) buffer < bufferEnd) {
        uint expectedValue = 0;

        for (int i = 0; i < groupSize; i++) {
            // overflow for uint is well defined
            expectedValue += hiprand(&state);
        }

        expectedValue *= multiplier;

        uint actualValue = *buffer;
        if (actualValue != expectedValue) {
            printf("Error found at byte offset %llu: expected %u but got %u\n", (char *) buffer - (char *) originalBuffer, expectedValue, actualValue);
            atomicAdd(errorCount, 1);
            // Only report one error per thread to avoid spamming prints
            break;
        }
        buffer += totalThreadCount;
    }
}

unsigned long long checkBuffer(void *ptr, int seed, size_t size, hipStream_t stream, int groupSize, int multiplier = 1) {
    unsigned long long *errorCount;
    CU_ASSERT(hipMalloc((hipDeviceptr_t *) &errorCount, sizeof(*errorCount)));
    CU_ASSERT(hipMemsetD8((hipDeviceptr_t) errorCount, 0, sizeof(*errorCount)));

    dim3 gridDim(NvLoom::getLocalMultiprocessorCount(), 1, 1);
    dim3 blockDim(numThreadPerBlock, 1, 1);
    patternCheckKernel<<<gridDim, blockDim, 0, stream>>>((uint *)ptr, seed, size, errorCount, groupSize, multiplier);
    CUDA_ASSERT(hipPeekAtLastError());
    CU_ASSERT(hipStreamSynchronize(stream));

    unsigned long long errorCountCopy;
    CU_ASSERT(cuMemcpy((hipDeviceptr_t) &errorCountCopy, (hipDeviceptr_t) errorCount, sizeof(errorCountCopy)));

    CU_ASSERT(hipFree((hipDeviceptr_t) errorCount));

    return errorCountCopy;
}

unsigned long long checkBuffer(void *ptr, int seed, size_t size, hipStream_t stream, CopyType copyType, int iterations) {
    if (copyType == COPY_TYPE_MULTICAST_LD_REDUCE) {
        return checkBuffer(ptr, seed, size, stream, MPIWrapper::getWorldSize(), 1);
    } else if (copyType == COPY_TYPE_MULTICAST_RED_ALL) {
        return checkBuffer(ptr, seed, size, stream, MPIWrapper::getWorldSize(), iterations);
    } else if (copyType == COPY_TYPE_MULTICAST_RED_SINGLE) {
        return checkBuffer(ptr, seed, size, stream, 1, iterations);
    } else {
        return checkBuffer(ptr, seed, size, stream, 1);
    }
}

void preloadKernels(int localDevice) {
    hipFuncAttributes unused;
    hipSetDevice(localDevice);
    hipFuncGetAttributes(&unused, reinterpret_cast<const void*>(&stridingMemcpyKernel<uint), write_to_multicast_memory<uint> >);
    hipFuncGetAttributes(&unused, reinterpret_cast<const void*>(&stridingMemcpyKernel<uint4), write_to_regular_memory<uint4> >);
    hipFuncGetAttributes(&unused, reinterpret_cast<const void*>(&simpleMemcpyKernel<uint), reduce_from_multicast_ld_reduce<uint> >);
    hipFuncGetAttributes(&unused, reinterpret_cast<const void*>(&simpleMemcpyKernel<uint), reduce_from_multicast_red<uint> >);
    hipFuncGetAttributes(&unused, reinterpret_cast<const void*>(&spinKernelDeviceMultistage));
    hipFuncGetAttributes(&unused, reinterpret_cast<const void*>(&patternFillKernel));
    hipFuncGetAttributes(&unused, reinterpret_cast<const void*>(&patternCheckKernel));
}
